#include<stdio.h>
#include"hip/hip_runtime.h"
#include<string.h>

#include<stdlib.h>


__global__ void cudacount(char *sentence,char *word,int *result,int sentencelenght,int wordlength){
    int idx=threadIdx.x;
    int flag=1;
    if(idx+wordlength<sentencelenght){
        for(int i=0;i<wordlength;i++){
            if(sentence[idx+i]!=word[i]){
                flag=0;
            }
        }if(flag)
        atomicAdd(result,1);
    }
}
int main(){
    char a[100]="hello helloooo,hello hellooooo,heooo";
    char b[6]="hello";
    int result=0;
    char *d_a,*d_b;
    int *d_c;
    int wordlen=strlen(a);
    hipMalloc((void**)&d_a,sizeof(char)*wordlen);
    hipMalloc((void**)&d_b,sizeof(char)*6);
    hipMalloc((void**)&d_c,sizeof(int));
    hipMemcpy(d_a,a,sizeof(char)*wordlen,hipMemcpyHostToDevice);
    hipMemcpy(d_b,b,sizeof(char)*6,hipMemcpyHostToDevice);
    hipMemcpy(d_c,&result,sizeof(int),hipMemcpyHostToDevice);
    cudacount<<<1,wordlen>>>(d_a,d_b,d_c,wordlen,6);
    hipMemcpy(&result,d_c,sizeof(int),hipMemcpyDeviceToHost);
    printf("%d",result);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}