#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>

__global__ void copyString(char* S, char* RS, int length) {
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;

    if (threadId < length * 4) {
        RS[threadId] = S[threadId % 4]; // Copy characters from S to RS
    }
}

int main() {
    const char* inputString = "PCAP"; // Input string
    int inputLength = strlen(inputString);

    // Allocate memory for S and RS on the host
    char* h_S = (char*)malloc(inputLength * sizeof(char));
    char* h_RS = (char*)malloc(inputLength * 4 * sizeof(char));

    // Copy input string from host to device
    char* d_S;
    char* d_RS;
    hipMalloc((void**)&d_S, inputLength * sizeof(char));
    hipMalloc((void**)&d_RS, inputLength * 4 * sizeof(char));
    hipMemcpy(d_S, inputString, inputLength * sizeof(char), hipMemcpyHostToDevice);

    // Launch the CUDA kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (inputLength * 4 + threadsPerBlock - 1) / threadsPerBlock;
    copyString<<<blocksPerGrid, threadsPerBlock>>>(d_S, d_RS, inputLength);

    // Copy the result from device to host
    hipMemcpy(h_RS, d_RS, inputLength * 4 * sizeof(char), hipMemcpyDeviceToHost);

    // Print the result
    printf("Input string S: %s\n", inputString);
    printf("Output string RS: %s\n", h_RS);

    // Cleanup
    free(h_S);
    free(h_RS);
    hipFree(d_S);
    hipFree(d_RS);

    return 0;
}
