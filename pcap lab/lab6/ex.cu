#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#define N 1024

__global__ void count(char* A,int *d_count){
	int i=threadIdx.x;
	if(A[i]=='a')
	atomicAdd(d_count,1);
}
int main(){
	char A[N];
	char *d_A;
	int count = 0, *d_count, result;
	printf("enter a string");
	fgets(A,N,stdin);
	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
	hipMalloc((void**)&d_A,strlen(A)*sizeof(char));
	hipMalloc((void**)&d_count,sizeof(int));
	hipMemcpy(d_A,A,strlen(A)*sizeof(char),hipMemcpyHostToDevice);
	hipMemcpy(d_count, &count, sizeof(int), hipMemcpyHostToDevice);
	 count<<<1, strlen(A)>>>(d_A, d_count);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	float time;
	hipEventElapsedTime(&time,start,stop);
	hipMemcpy(&result,d_count,sizeof(int),hipMemcpyDeviceToHost);
	printf("count=%d\n",result);
	printf("time=%f\n",time);
	hipFree(d_A);
	hipFree(d_count);
}


