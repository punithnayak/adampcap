#include"hip/hip_runtime.h"

#include<stdio.h>
#include<stdlib.h>


__global__ void addmatrix(int *a,int *b,int *c){
	int n=threadIdx.x,m=blockIdx.x,size=blockDim.x;
	c[m*size+n]=a[m*size+n]+b[m*size+n];

	}

int main()
{
	int *a,*b,*c,m,n;
	int *d_a,*d_b,*d_c;
	printf("enter the value of m and n\n");
	scanf("%d",&m);
	scanf("%d",&n);
	int size=sizeof(int)*m*n;
	a=(int*)malloc(size);
	b=(int*)malloc(size);
	c=(int*)malloc(size);
    printf("enter matrix1:");
    for (int i = 0; i < m*n; ++i)
    {
    	scanf("%d",&a[i]);
    }
    printf("enter matrix2:");
    for (int i = 0; i < m*n; ++i)
    {
    	scanf("%d",&b[i]);
    }
	hipMalloc((void**)&d_a,size);
	hipMalloc((void**)&d_b,size);
	hipMalloc((void**)&d_c,size);
	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);
	addmatrix<<<m,n>>>(d_a,d_b,d_c);
	hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);
	printf("result vector\n");
	for (int i = 0; i < m; i++)
	{
		for (int j = 0; j < n; j++)
		{
			printf("%d ",c[i*n+j] );
		}printf("\n");
	}
	hipFree(d_a);
	hipFree(d_b);
	return 0;

}