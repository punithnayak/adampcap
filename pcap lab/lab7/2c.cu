#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

__global__ void multiply_elementwise(int *a, int *b, int *c, int wa, int wb) {
    int rida = threadIdx.y;
    int cidb = threadIdx.x;
    int sum = 0;
    for (int k = 0; k < wa; k++)
        sum += a[rida * wa + k] * b[k * wb + cidb];
    c[rida * wb + cidb] = sum;
}

int main() {
    int *a, *b, *c, wa, ha, wb, hb;
    int *d_a, *d_b, *d_c;
    
    printf("Enter value of wa, ha:\n");
    scanf("%d %d", &wa, &ha);
    
    printf("Enter value of wb, hb:\n");
    scanf("%d %d", &wb, &hb);

    if (wa != hb) {
        printf("Matrix dimensions are not compatible for multiplication.\n");
        return 1;
    }

    int size1 = sizeof(int) * wa * ha;
    int size2 = sizeof(int) * wb * hb;

    a = (int*)malloc(size1);
    b = (int*)malloc(size2);
    c = (int*)malloc(sizeof(int) * wa * wb);

    hipMalloc((void**)&d_a, size1);
    hipMalloc((void**)&d_b, size2);
    hipMalloc((void**)&d_c, sizeof(int) * wa * wb);

    printf("Enter matrix1:\n");
    for (int i = 0; i < wa * ha; i++)
        scanf("%d", &a[i]);

    printf("Enter matrix2:\n");
    for (int i = 0; i < wb * hb; i++)
        scanf("%d", &b[i]);

    hipMemcpy(d_a, a, size1, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size2, hipMemcpyHostToDevice);

    dim3 blockDims(wb, ha);
    multiply_elementwise<<<1, blockDims>>>(d_a, d_b, d_c, wa, wb);

    hipMemcpy(c, d_c, sizeof(int) * wa * wb, hipMemcpyDeviceToHost);

    printf("Resultant matrix:\n");
    for (int i = 0; i < ha; i++) {
        for (int j = 0; j < wb; j++)
            printf("%d ", c[i * wb + j]);
        printf("\n");
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    free(a);
    free(b);
    free(c);

    return 0;
}
