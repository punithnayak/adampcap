#include"hip/hip_runtime.h"

#include<stdio.h>
#include<stdlib.h>

__global__ void multiply_rowwise(int *a,int *b,int *c,int wa,int wb){
	int rida=threadIdx.x;
	int sum;
	for(int cidb=0;cidb<wb;cidb++){
	sum=0;
	for(int k=0;k<wa;k++)
	sum+=a[rida*wa+k]*b[k*wb+cidb];
	c[rida*wb+cidb]=sum;
	}
}
int main(){
	int *a,*b,*c,wa,ha,wb,hb;
	int *d_a,*d_b,*d_c;
	printf("enter valude of wa,ha\n");
	scanf("%d",&wa);
	scanf("%d",&ha);
	printf("enter valude of wb,hb\n");
	scanf("%d",&wb);
	scanf("%d",&hb);
	int size1=sizeof(int)*wa*ha,size2=sizeof(int)*wb*hb;
	a=(int*)malloc(size1);
	b=(int*)malloc(size2);
	c=(int*)malloc(sizeof(int)*wa*hb);
	hipMalloc((void**)&d_a,size1);
	hipMalloc((void**)&d_b,size2);
	hipMalloc((void**)&d_c,sizeof(int)*wa*hb);
	printf("enter matrix1\n");
	for(int i=0;i<wa*ha;i++)
	scanf("%d",&a[i]);
	printf("enter matrix2\n");
	for(int i=0;i<wb*hb;i++)
	scanf("%d",&b[i]);
	hipMemcpy(d_a,a,size1,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,size2,hipMemcpyHostToDevice);
    multiply_rowwise<<<1,ha>>>(d_a,d_b,d_c,wa,wb);
    hipMemcpy(c,d_c,sizeof(int)*wa*hb,hipMemcpyDeviceToHost);
    printf("resultant matrix\n");
    for(int i=0;i<wa;i++){
    for(int j=0;j<hb;j++)
    printf("%d ",c[i*hb+j]);
    printf("\n");
    }
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);


}
