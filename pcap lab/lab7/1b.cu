#include"hip/hip_runtime.h"

#include<stdio.h>
#include<stdlib.h>


__global__ void addmatrix(int *a,int *b,int *c,int m,int n){
	int i=threadIdx.x;
	if(i<n){
		for (int j = 0; j < m; j++)
		{
			c[j*n+i]=a[j*n+i]+b[j*n+i];
		}
	}
}

int main()
{
	int *a,*b,*c,m,n;
	int *d_a,*d_b,*d_c;
	printf("enter the value of m and n\n");
	scanf("%d",&m);
	scanf("%d",&n);
	int size=sizeof(int)*m*n;
	a=(int*)malloc(size);
	b=(int*)malloc(size);
	c=(int*)malloc(size);
    printf("enter matrix1:");
    for (int i = 0; i < m*n; ++i)
    {
    	scanf("%d",&a[i]);
    }
    printf("enter matrix2:");
    for (int i = 0; i < m*n; ++i)
    {
    	scanf("%d",&b[i]);
    }
	hipMalloc((void**)&d_a,size);
	hipMalloc((void**)&d_b,size);
	hipMalloc((void**)&d_c,size);
	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);
	addmatrix<<<1,n>>>(d_a,d_b,d_c,m,n);
	hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);
	printf("result vector\n");
	for (int i = 0; i < m; i++)
	{
		for (int j = 0; j < n; j++)
		{
			printf("%d ",c[i*n+j] );
		}printf("\n");
	}
	hipFree(d_a);
	hipFree(d_b);
	return 0;

}
