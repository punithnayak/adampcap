#include<hip/hip_runtime.h>

#include<stdio.h>
#include<stdlib.h>
__global__ void odd(int *a){
	int idx=threadIdx.x;
	int n=blockDim.x;
	if((idx%2)!=0&& (idx+1)<n){
	if(a[idx]>a[idx+1]){
	int temp=a[idx];
	a[idx]=a[idx+1];
	a[idx+1]=temp;
	}
	}
}
__global__ void even(int *a){
	int idx=threadIdx.x;
	int n=blockDim.x;
	if((idx%2)==0&& (idx+1)<n){
	if(a[idx]>a[idx+1]){
	int temp=a[idx];
	a[idx]=a[idx+1];
	a[idx+1]=temp;
	}
	}
}


int main(){
	int *a;
	int *d_a;
	int n;
	printf("enter the size of the array");
	scanf("%d",&n);
	a=(int*)malloc(sizeof(int)*n);
	printf("enter the array");
	for(int i=0;i<n;i++)
	scanf("%d",&a[i]);
	hipMalloc((void**)&d_a,sizeof(int)*n);
	hipMemcpy(d_a,a,sizeof(int)*n,hipMemcpyHostToDevice);
	for(int i=0;i<n/2;i++){
	odd<<<1,n>>>(d_a);
	even<<<1,n>>>(d_a);
	}
	hipMemcpy(a,d_a,sizeof(int)*n,hipMemcpyDeviceToHost);
	for(int i=0;i<n;i++)
	printf("%d ",a[i]);
	hipFree(d_a);


}
