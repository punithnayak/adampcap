
#include <hip/hip_runtime.h>
#include <stdio.h>

// Define the filter size and constant memory size
#define FILTER_SIZE 3
#define DATA_SIZE 10

// Define the filter in constant memory
__constant__ int filter[FILTER_SIZE];

__global__ void convolution(int* input, int* output) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int halfFilterSize = FILTER_SIZE / 2;
    int result = 0;

    for (int i = 0; i < FILTER_SIZE; i++) {
        int inputIndex = tid - halfFilterSize + i;
        if (inputIndex >= 0 && inputIndex < DATA_SIZE) {
            result += input[inputIndex] * filter[i];
        }
    }

    output[tid] = result;
}

int main() {
    int data[DATA_SIZE];
    int result[DATA_SIZE];
    int hostFilter[FILTER_SIZE] = {1, 2, 1};  // Define the filter on the host

    // Initialize data and filter
    for (int i = 0; i < DATA_SIZE; i++) {
        data[i] = i;
    }

    // Copy the filter from the host to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(filter), hostFilter, FILTER_SIZE * sizeof(int));

    int* d_data;
    int* d_result;
    hipMalloc((void**)&d_data, DATA_SIZE * sizeof(int));
    hipMalloc((void**)&d_result, DATA_SIZE * sizeof(int));

    // Copy data to the device
    hipMemcpy(d_data, data, DATA_SIZE * sizeof(int), hipMemcpyHostToDevice);

    // Configure kernel launch
    int block_size = 256;
    int grid_size = (DATA_SIZE + block_size - 1) / block_size;

    convolution<<<grid_size, block_size>>>(d_data, d_result);

    // Copy the result back to the host
    hipMemcpy(result, d_result, DATA_SIZE * sizeof(int), hipMemcpyDeviceToHost);

    // Print the result
    for (int i = 0; i < DATA_SIZE; i++) {
        printf("%d\n", result[i]);
    }

    hipFree(d_data);
    hipFree(d_result);

    return 0;
}
