#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include<stdlib.h>

__device__ int co_rank(long int k, long int* A, long int m, long int* B, long int n)
{
	int i = k < m ? k : m;
	int j = k - i;
	int i_low = 0 > (k - n) ? 0 : (k - n);
	int j_low = 0 > (k - m) ? 0 : (k - m);
	int delta;
	bool active = true;
	while (active)
	{
		if (i > 0 && j<n && A[i - 1]>B[j])
		{
			delta = ((i - i_low+1)>>1);
			j_low = j;
			j = j + delta;
			i = i - delta;
		}
		else if (j > 0 && i < m && B[j - 1] >= A[i])
		{
			delta = ((j - j_low+1)>>2);
			i_low = i;
			j = j - delta;
			i = i + delta;

		}
		else
			active = false;
	}
	return i;
}
__device__ void merge_sequential(long int* A, long int m, long int* B,long int n, long int* C)
{
	int i = 0;
	int j = 0;
	int k = 0;
	while ((i < m) && (j < n))
	{
		if (A[i] <= B[j])
		{
			C[k++] = A[i++];
		}
		else
		{
			C[k++] = B[j++];
		}
	}
	if (i == m)
	{
		for (; j < n; j++)
			C[k++] = B[j];
	}
	else
	{
		for (; i < m; i++)
			C[k++] = A[i];
	}
}
__global__ void merge_basic(long int* A,long int m, long int *B, long int n, long int *C)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int k_current = idx * ((m + n)+ (blockDim.x * gridDim.x )/ (blockDim.x * gridDim.x));
	int k_next = (idx + 1) * (((m + n) + (blockDim.x * gridDim.x)) / (blockDim.x * gridDim.x)) < (m + n)?  ((m + n) + (blockDim.x * gridDim.x) / (blockDim.x * gridDim.x )):(m+n);
	int i_current = co_rank(k_current, A, m, B, n);
	int i_next = co_rank(k_next, A, m, B, n);
	int j_current = k_current - i_current;
	int j_next = k_next - i_next;
	merge_sequential(&A[i_current], i_next - i_current, &B[j_current], j_next - j_current, &C[k_current]);
}

int main()
{
	long int* dev_a = 0;
	long int* dev_b = 0;
	long int* dev_c = 0;
	long int* a, *b, *c;
	long int size, size2;
	printf("Enter the size of the first array");
	scanf("%ld", &size);
	a = (long int*)malloc(sizeof(long int) * size);
	printf("Enter the array");
	for (long int i = 0; i < size; i++)
	{
		scanf("%ld",&a[i]);
	}
	printf("Enter the size of the second array");
	scanf("%ld", &size2);
	b = (long int*)malloc(sizeof(long int) * size2);
	printf("Enter the array");
	for (long int i = 0; i < size2; i++)
	{
		scanf("%ld", &b[i]);
	}
	c = (long int*)malloc(sizeof(long int) * (size+size2));
	hipMalloc((void**)&dev_a, size * sizeof(long int));
	hipMalloc((void**)&dev_b, size2 * sizeof(long int));
	hipMalloc((void**)&dev_c, (size+size2) * sizeof(long int));
	hipMemcpy(dev_a, a, size * sizeof(long int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, size2 * sizeof(long int), hipMemcpyHostToDevice);

	

		merge_basic << < 1, 3 >> > (dev_a,size,dev_b,size2,dev_c);
		

	
	hipMemcpy(c, dev_c, (size+size2) * sizeof(long int), hipMemcpyDeviceToHost);
	printf("Result\n");
	for (long int w = 0; w < (size+size2); w++)
	{
		printf("%ld\t", c[w]);
	}
	hipFree(dev_a);
	return 0;
}