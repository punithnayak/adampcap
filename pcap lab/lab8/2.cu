#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void modifyMatrix(int *matrix, int rows, int cols) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < rows) {
        if (j < cols) {
            matrix[i * cols + j] = pow(matrix[i * cols + j] , (i + 1));
        }
    }
}

int main() {
    int M, N;

    printf("Enter the number of rows (M) and columns (N) of the matrix: ");
    scanf("%d %d", &M, &N);

    int *matrix;
    int size = M * N * sizeof(int);

    matrix = (int*)malloc(size);

    int *d_matrix;
    hipMalloc((void**)&d_matrix, size);

    printf("Enter matrix elements (%dx%d):\n", M, N);
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            scanf("%d", &matrix[i * N + j]);
        }
    }

    hipMemcpy(d_matrix, matrix, size, hipMemcpyHostToDevice);

    dim3 blockDim(16, 16);
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x, (M + blockDim.y - 1) / blockDim.y);

    modifyMatrix<<<gridDim, blockDim>>>(d_matrix, M, N);

    hipMemcpy(matrix, d_matrix, size, hipMemcpyDeviceToHost);

    printf("Modified Matrix:\n");
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            printf("%d ", matrix[i * N + j]);
        }
        printf("\n");
    }

    free(matrix);
    hipFree(d_matrix);

    return 0;
}
