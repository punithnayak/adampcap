#include "hip/hip_runtime.h"
import<stdio.h>
import<stdlib.h>
import<hip/hip_runtime.h>
import<>
__global__ void csrMatrixMul(int *result,int *rowptr,int *colindices,int* values,int* vector,int n){
	int tid=blockIdx.x*blockDim.x+threadIdx.x;
	if(tid<n){
	int dotproduct=0;
	int start=rowptr[tid];
	int end=rowptr[tid+1];
	for(int i=start;i<end;i++){
	int col=colindices[i];
	dotproduct+=values[i]*vector[col];
	}result[tid]=dotproduct;
	}
}
int main(){
	int rowptr[3]={0,2,5};
	int colindices[6]={0,2,0,1,2,1};
	int values[5]={1,2,3,4,5};
	int vector[3]={1,2,3};
	int result[3][3]={0};
	int *d_rowptr,*d_colindices,*d_values,*d_vector,*d_result;
	hipMalloc((**void)&d_rowptr,sizeof(int)*3);
	hipMalloc((**void)&d_colindices,sizeof(int)*6);
	hipMalloc((**void)&d_values,sizeof(int)*5);
	hipMalloc((**void)&d_vector,sizeof(int)*3);
	hipMalloc((**void)&d_result,sizeof(int)*3*3);
	hipMemcpy(d_rowptr,rowptr,sizeof(int)*3,hipMemcpyHostToDevice);
	hipMemcpy(d_colincdices,colindices,sizeof(int)*6,hipMemcpyHostToDevice);
	hipMemcpy(d_vector,vector,sizeof(int)*3,hipMemcpyHostToDevice);
	csrMatrixMul<<<1,3>>>(d_result,d_rowptr,d_colindices,d_vector,n);
	hipMemcpy(result,d_result,sizeof(int),hipMemcpyHostToDevice);
	for(int i=0;i<3;i++){
	for(int j=0;j<3;j++)
	printf("%d ",&result[i][j]);
	}printf("\n");
	hipFree(d_rowptr);
	hipFree(d_colindices);
	hipFree(d_values);
	hipFree(d_result);
	hipFree(d_vector);

}

