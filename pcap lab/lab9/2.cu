#include <stdio.h>
#include <hip/hip_runtime.h>

#define MASK_SIZE 3 


__global__ void convolution2D(int *input, int *output, int *mask, int width, int height) {
    int tx=threadIdx.x;
    int ty=threadIdx.y;
    int col=blockDim.x*blockIdx.x+tx;
    int row=blockDim.y*blockIdx.y+ty;
    int halfmask=MASK_SIZE/2;
    int sum=0;
    for(int i=-halfmask;i<=halfmask;i++){
    for(int j=-halfmask;j<=halfmask;j++){
    int inputX=col+i;
    int inputY=row+j;
    if(inputX>=0&&inputX<width&&inputY>=0&&inputY<height)
    sum+=input[inputX*width+inputY]*mask[(i+halfmask)*MASK_SIZE+(j+halfmask)];
    }
    }
    output[row*width+col]=sum;
}

int main() {
    int width = 4; 
    int height = 4;
    
    int input[] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16}; 
    int mask[] = {1, 2, 1, 0, 0, 0, -1, -2, -1}; 
    int output[width * height]; 

    int *d_input, *d_output, *d_mask;
    
    hipMalloc((void**)&d_input, sizeof(int) * width * height);
    hipMalloc((void**)&d_output, sizeof(int) * width * height);
    hipMalloc((void**)&d_mask, sizeof(int) * MASK_SIZE * MASK_SIZE);

  
    hipMemcpy(d_input, input, sizeof(int) * width * height, hipMemcpyHostToDevice);
    hipMemcpy(d_mask, mask, sizeof(int) * MASK_SIZE * MASK_SIZE, hipMemcpyHostToDevice);

    dim3 gridSize(width, height);
    dim3 blockSize(MASK_SIZE, MASK_SIZE);

    convolution2D<<<gridSize, blockSize>>>(d_input, d_output, d_mask, width, height);

    hipMemcpy(output, d_output, sizeof(int) * width * height, hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_mask);

   
    printf("Input Image:\n");
    for (int i = 0; i < height; i++) {
        for (int j = 0; j < width; j++) {
            printf("%d ", input[i * width + j]);
        }
        printf("\n");
    }

    printf("\nConvolution Mask:\n");
    for (int i = 0; i < MASK_SIZE; i++) {
        for (int j = 0; j < MASK_SIZE; j++) {
            printf("%d ", mask[i * MASK_SIZE + j]);
        }
        printf("\n");
    }

    printf("\nOutput Image (after 2D convolution):\n");
    for (int i = 0; i < height; i++) {
        for (int j = 0; j < width; j++) {
            printf("%d ", output[i * width + j]);
        }
        printf("\n");
    }

    return 0;
}
