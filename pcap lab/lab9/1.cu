#include<stdio.h>
#include"hip/hip_runtime.h"

__global__ void matrixMul(int *a,int *b,int *c,int m1,int n1,int n2){
	int col=threadIdx.x+blockIdx.x*blockDim.x;
	int row=threadIdx.y+blockIdx.y*blockDim.y;
	if(row<m1 && col<n2){
	int sum=0;
	for (int k = 0; k < n1; k++) {
            sum += a[row * n1 + k] * b[k * n2 + col];
    }
        c[row * n2 + col] = sum;
	}

}
int main() {
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    int m1, n1;
    int m2, n2;
    
    printf("Enter the value of m1 and n1\n");
    scanf("%d %d", &m1, &n1);
    
    printf("Enter the value of m2 and n2\n");
    scanf("%d %d", &m2, &n2);
    
    a = (int*)malloc(sizeof(int) * m1 * n1);
    b = (int*)malloc(sizeof(int) * m2 * n2);
    c = (int*)malloc(sizeof(int) * m1 * n2);
    
    hipMalloc((void**)&d_a, sizeof(int) * m1 * n1);
    hipMalloc((void**)&d_b, sizeof(int) * m2 * n2);
    hipMalloc((void**)&d_c, sizeof(int) * m1 * n2);
    
    printf("Enter matrix 1\n");
    for (int i = 0; i < m1 * n1; i++) {
        scanf("%d", &a[i]);
    }
    
    printf("Enter matrix 2\n");
    for (int i = 0; i < m2 * n2; i++) {
        scanf("%d", &b[i]);
    }
    
    hipMemcpy(d_a, a, sizeof(int) * m1 * n1, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(int) * m2 * n2, hipMemcpyHostToDevice);
    
    dim3 dimGrid((n2 + 4) / 4, (m1 + 4) / 4);
    dim3 blockGrid(2, 2);
    matrixMul<<<dimGrid, blockGrid>>>(d_a, d_b, d_c, m1, n1, n2);
    
    hipMemcpy(c, d_c, sizeof(int) * m1 * n2, hipMemcpyDeviceToHost);
    
    printf("Resultant matrix\n");
    for (int i = 0; i < m1; i++) {
        for (int j = 0; j < n2; j++) {
            printf("%d ", c[i * n2 + j]);
        }
        printf("\n");
    }
    
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(a);
    free(b);
    free(c);

    return 0;
}
