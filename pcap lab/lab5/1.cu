#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void vecAdd(int *a,int *b,int *c,int n){
	int idx=blockDim.x*blockIdx.x+threadIdx.x;
	if(idx<n){
	c[idx]=a[idx]+b[idx];
	}
}
int main(){
	int ha[8]={1,1,1,1,1,1,1,1};
	int hb[8]={1,1,1,1,1,1,1,1};
	int hc[8];
	int *da,*db,*dc;
	int size=sizeof(int);
	hipMalloc(&da,size*8);
	hipMalloc(&db,size*8);
	hipMalloc(&dc,size*8);
	hipMemcpy(da,ha,size*8,hipMemcpyHostToDevice);
	hipMemcpy(db,hb,size*8,hipMemcpyHostToDevice);
	vecAdd<<<1,32>>>(da,db,dc,8);
	hipMemcpy(hc,dc,size*8,hipMemcpyDeviceToHost);
	for(int i=0;i<8;i++)
		printf("%d ",hc[i]);
	hipFree(da);
	hipFree(db);
	hipFree(dc);
	return 0;

}