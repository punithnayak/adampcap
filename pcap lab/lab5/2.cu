#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

__global__ void vecAdd(int *a, int *b, int *c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}
int main() {
    int N = 1024;
    int ha[N], hb[N], hc[N];  /
    int *da, *db, *dc;  
    int size = N * sizeof(int);
    for (int i = 0; i < N; i++) {
        ha[i] = 1;
        hb[i] = 1;
    }
    hipMalloc((void**)&da, size);
    hipMalloc((void**)&db, size);
    hipMalloc((void**)&dc, size);

    hipMemcpy(da, ha, size, hipMemcpyHostToDevice);
    hipMemcpy(db, hb, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vecAdd<<<blocksPerGrid, threadsPerBlock>>>(da, db, dc, N);
    hipMemcpy(hc, dc, size, hipMemcpyDeviceToHost);
    for (int i = 0; i < N; i++) {
        printf("%d ", hc[i]);
    }
    hipFree(da);
    hipFree(db);
    hipFree(dc);
    return 0;
}
