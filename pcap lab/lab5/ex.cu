#include "hip/hip_runtime.h"

#include<stdio.h>
__global__ void add(int *a,int *b,int *c){
	*c=*a+*b;
}
int main(void){
	int a,b,c;\
	int *da,*db,*dc;
	int size=sizeof(int);
	hipMalloc((void**)&da,size);
	hipMalloc((void**)&db,size);
	hipMalloc((void**)&dc,size);
	a=5;
	b=6;
	hipMemcpy(da,&a,size,hipMemcpyHostToDevice);
	hipMemcpy(db,&b,size,hipMemcpyHostToDevice);
	add<<<1,1>>>(da,db,dc);
	hipMemcpy(&c,dc,size,hipMemcpyDeviceToHost);
	printf("%d",c);
	hipFree(da);
	hipFree(db);
	hipFree(dc);
	return 0;
}