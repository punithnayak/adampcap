#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void csin(float *a,float *b,int n){
	int idx=blockIdx.x;
	if(idx<n){
	b[idx]=sinf(a[idx]);
	}
}
int main(){
	float a[6]={0,0.3,0.45,.6,.9,1};
	float b[6];
	int size=sizeof(float);
	float *da,*db;
	hipMalloc((void**)&da,size*6);
	hipMalloc((void**)&db,size*6);
	
	hipMemcpy(da,a,size*6,hipMemcpyHostToDevice);

	csin<<<6,1>>>(da,db,6);
	hipMemcpy(b,db,size*6,hipMemcpyDeviceToHost);
	for(int i=0;i<6;i++)
		printf("%f ",b[i]);
	hipFree(da);
	hipFree(db);
	return 0;

}